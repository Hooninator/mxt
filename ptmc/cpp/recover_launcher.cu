#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include "recover_kernel.cuh"

void print_tensor(const torch::Tensor& t, const std::string& name = "") {
    if (!name.empty()) std::cout << name << ": ";
    
    // Move to CPU if on GPU
    auto cpu_tensor = t.cpu();
    
    std::cout << "shape=" << cpu_tensor.sizes() << " dtype=" << cpu_tensor.dtype() << "\n";
    
    // Print values based on dtype
    if (cpu_tensor.dtype() == torch::kFloat32) {
        auto data = cpu_tensor.data_ptr<float>();
        for (int64_t i = 0; i < cpu_tensor.numel(); ++i) {
            std::cout << data[i] << " ";
            if ((i + 1) % 10 == 0) std::cout << "\n";
        }
    } else if (cpu_tensor.dtype() == torch::kFloat64) {
        auto data = cpu_tensor.data_ptr<double>();
        for (int64_t i = 0; i < cpu_tensor.numel(); ++i) {
            std::cout << data[i] << " ";
            if ((i + 1) % 10 == 0) std::cout << "\n";
        }
    } else if (cpu_tensor.dtype() == torch::kHalf) {
        auto data = cpu_tensor.data_ptr<at::Half>();
        for (int64_t i = 0; i < cpu_tensor.numel(); ++i) {
            std::cout << (float)data[i] << " ";
            if ((i + 1) % 10 == 0) std::cout << "\n";
        }
    }
    std::cout << "\n\n";
}

template<typename T>
void print_device_ptr(const T* d_ptr, int64_t count, const std::string& name = "") {
    if (!name.empty()) std::cout << name << ": ";
    
    // Copy to host
    std::vector<T> host_data(count);
    CUDA_CHECK(hipMemcpy(host_data.data(), d_ptr, sizeof(T) * count, hipMemcpyDeviceToHost));
    
    for (int64_t i = 0; i < count; ++i) {
        std::cout << host_data[i] << " ";
        if ((i + 1) % 10 == 0) std::cout << "\n";
    }
    std::cout << "\n\n";
}

template <typename InT, typename OutT, typename AccT, bool UseF64=false>
void launch_diag_unscale2(
    const at::Tensor& tY, at::Tensor& Y, const double theta,
    const std::vector<int64_t>& sizes, const int64_t sizes_prod,
    const std::vector<at::Tensor>& r_inv_vecs)
{
    const int N = (int)sizes.size();
    auto tY_ptr = tY.data_ptr<InT>();
    auto Y_ptr  = Y.data_ptr<OutT>();

    // Pack sizes/strides to device
    at::Tensor sizes_t   = torch::empty({N}, torch::dtype(torch::kInt64).device(tY.device()));
    CUDA_CHECK(hipMemcpyAsync(sizes_t.data_ptr<int64_t>(), sizes.data(),   sizeof(int64_t)*N, hipMemcpyHostToDevice));

    // Build array of r_inv pointers 
    std::vector<const AccT*> r_inv_host(N, nullptr);
    for (int d = 0; d < N; ++d) {
        r_inv_host[d] = r_inv_vecs[d].data_ptr<AccT>();
    }
    // Copy pointer array to device
    const AccT** d_r_inv = nullptr;
    CUDA_CHECK(hipMalloc(&d_r_inv, sizeof(AccT*)*N));
    CUDA_CHECK(hipMemcpyAsync(d_r_inv, r_inv_host.data(), sizeof(AccT*)*N, hipMemcpyHostToDevice));


    // grid
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);

    const uint32_t blocks_per_sm = 16;

    const uint32_t block = 1024;
    const uint32_t grid = prop.multiProcessorCount * blocks_per_sm;

    using KernelInT = std::conditional_t<std::is_same_v<InT, at::Half>, half, InT>;
    using KernelOutT = std::conditional_t<std::is_same_v<OutT, at::Half>, half, OutT>;
    using KernelAccT = std::conditional_t<std::is_same_v<AccT, at::Half>, half, AccT>;

    // Cast pointers directly for kernel calls
    auto cast_input_ptr = [&]() -> const KernelInT* {
        if constexpr (std::is_same_v<InT, at::Half>) {
            return reinterpret_cast<const half*>(tY_ptr);
        } else {
            return tY_ptr;
        }
    };
    
    auto cast_output_ptr = [&]() -> KernelOutT* {
        if constexpr (std::is_same_v<OutT, at::Half>) {
            return reinterpret_cast<half*>(Y_ptr);
        } else {
            return Y_ptr;
        }
    };
    
    auto cast_r_inv_ptr = [&]() -> const KernelAccT** {
        if constexpr (std::is_same_v<AccT, at::Half>) {
            return reinterpret_cast<const half**>(d_r_inv);
        } else {
            return d_r_inv;
        }
    };

    CUDA_CHECK(hipDeviceSynchronize());
    switch (N) {
      case 2: diag_unscale_kernel2<2,KernelInT,KernelOutT,KernelAccT,UseF64><<<grid,block>>>(
                  cast_input_ptr(), cast_output_ptr(), (KernelOutT)theta,
                  sizes_t.data_ptr<int64_t>(), sizes_prod, 
                  cast_r_inv_ptr()); break;
      case 3: diag_unscale_kernel2<3,KernelInT,KernelOutT,KernelAccT,UseF64><<<grid,block>>>(
                  cast_input_ptr(), cast_output_ptr(), (KernelOutT)theta,
                  sizes_t.data_ptr<int64_t>(), sizes_prod, 
                  cast_r_inv_ptr()); break;
      case 4: diag_unscale_kernel2<4,KernelInT,KernelOutT,KernelAccT,UseF64><<<grid,block>>>(
                  cast_input_ptr(), cast_output_ptr(), (KernelOutT)theta, 
                  sizes_t.data_ptr<int64_t>(), sizes_prod, 
                  cast_r_inv_ptr()); break;
    }

    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipFree(d_r_inv));
}


template <typename InT, typename OutT, typename AccT, bool UseF64=false>
void launch_diag_unscale(
    const at::Tensor& tY, at::Tensor& Y, const double theta,
    const std::vector<int64_t>& sizes, const std::vector<int64_t>& strides_elts,
    const std::vector<at::Tensor>& r_inv_vecs)
{
    constexpr int NMAX = 16;
    const int N = (int)sizes.size();
    TORCH_CHECK(N >= 1 && N <= NMAX);
    TORCH_CHECK(N >= 2, "This fast kernel expects N>=2 (has 2D tiling).");

    // Device pointers
    auto tY_ptr = tY.data_ptr<InT>();
    auto Y_ptr  = Y.data_ptr<OutT>();

    // Pack sizes/strides to device
    at::Tensor sizes_t   = torch::empty({N}, torch::dtype(torch::kInt64).device(tY.device()));
    at::Tensor strides_t = torch::empty({N}, torch::dtype(torch::kInt64).device(tY.device()));
    CUDA_CHECK(hipMemcpyAsync(sizes_t.data_ptr<int64_t>(), sizes.data(),   sizeof(int64_t)*N, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpyAsync(strides_t.data_ptr<int64_t>(), strides_elts.data(), sizeof(int64_t)*N, hipMemcpyHostToDevice));

    // Build array of r_inv pointers 
    std::vector<const AccT*> r_inv_host(N, nullptr);
    for (int d = 0; d < N; ++d) {
        r_inv_host[d] = r_inv_vecs[d].data_ptr<AccT>();
    }
    // Copy pointer array to device
    const AccT** d_r_inv = nullptr;
    CUDA_CHECK(hipMalloc(&d_r_inv, sizeof(AccT*)*N));
    CUDA_CHECK(hipMemcpyAsync(d_r_inv, r_inv_host.data(), sizeof(AccT*)*N, hipMemcpyHostToDevice));

    // Grid
    const int64_t rows = sizes[N-2];
    const int64_t cols = sizes[N-1];
    dim3 block(32, 16, 1);  // 512 threads total; optimized for register pressure and occupancy
    dim3 grid(
        (cols + TILE_COLS - 1) / TILE_COLS,
        (rows + TILE_ROWS - 1) / TILE_ROWS,
        std::max<int64_t>(1, std::accumulate(sizes.begin(), sizes.end()-2, int64_t(1), std::multiplies<int64_t>()))
    );
    // Clamp grid.z if huge: loop over outer tiles inside the kernel (left as an exercise)
    TORCH_CHECK(grid.z <= 2147483647, "Outer grid too large; implement looping over outer tiles.");

    // Dispatch by N with a switch for full unrolling of small loops
    // Use __half for kernel template parameters, cast pointers appropriately
    using KernelInT = std::conditional_t<std::is_same_v<InT, at::Half>, half, InT>;
    using KernelOutT = std::conditional_t<std::is_same_v<OutT, at::Half>, half, OutT>;
    using KernelAccT = std::conditional_t<std::is_same_v<AccT, at::Half>, half, AccT>;
    
    // Cast pointers directly for kernel calls
    auto cast_input_ptr = [&]() -> const KernelInT* {
        if constexpr (std::is_same_v<InT, at::Half>) {
            return reinterpret_cast<const half*>(tY_ptr);
        } else {
            return tY_ptr;
        }
    };
    
    auto cast_output_ptr = [&]() -> KernelOutT* {
        if constexpr (std::is_same_v<OutT, at::Half>) {
            return reinterpret_cast<half*>(Y_ptr);
        } else {
            return Y_ptr;
        }
    };
    
    auto cast_r_inv_ptr = [&]() -> const KernelAccT** {
        if constexpr (std::is_same_v<AccT, at::Half>) {
            return reinterpret_cast<const half**>(d_r_inv);
        } else {
            return d_r_inv;
        }
    };
    
    switch (N) {
      case 2: diag_unscale_kernel<2,KernelInT,KernelOutT,KernelAccT,UseF64><<<grid,block>>>(
                  cast_input_ptr(), cast_output_ptr(), theta,
                  sizes_t.data_ptr<int64_t>(), strides_t.data_ptr<int64_t>(),
                  cast_r_inv_ptr(), grid.z); break;
      case 3: diag_unscale_kernel<3,KernelInT,KernelOutT,KernelAccT,UseF64><<<grid,block>>>(
                  cast_input_ptr(), cast_output_ptr(), theta,
                  sizes_t.data_ptr<int64_t>(), strides_t.data_ptr<int64_t>(),
                  cast_r_inv_ptr(), grid.z); break;
      case 4: diag_unscale_kernel<4,KernelInT,KernelOutT,KernelAccT,UseF64><<<grid,block>>>(
                  cast_input_ptr(), cast_output_ptr(), theta, 
                  sizes_t.data_ptr<int64_t>(), strides_t.data_ptr<int64_t>(), 
                  cast_r_inv_ptr(), grid.z); break;
      default: diag_unscale_kernel<8,KernelInT,KernelOutT,KernelAccT,UseF64><<<grid,block>>>(
                  cast_input_ptr(), cast_output_ptr(), theta, 
                  sizes_t.data_ptr<int64_t>(), strides_t.data_ptr<int64_t>(), 
                  cast_r_inv_ptr(), grid.z); break;
    }
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipFree(d_r_inv));
}

torch::Tensor diag_unscale_forward2(
    torch::Tensor tY,                 // \tilde{Y} (contiguous recommended)
    std::vector<torch::Tensor> r_inv, // length N, each 1D on device
    double theta,                     // scalar used to scale output tensor
    bool use_fp64_accum,              // Accumulate in FP64?
    c10::ScalarType out_dtype         // kFloat or kDouble, etc.
)
{
    const int N = tY.dim();
    TORCH_CHECK((int)r_inv.size() == N);

    // Build sizes and strides in "elements"
    std::vector<int64_t> sizes(N);
    for (int d = 0; d < N; ++d) 
    {
        sizes[d] = tY.size(d);
    }

    auto sizes_prod = std::reduce(sizes.begin(), sizes.end(), 1, std::multiplies<int64_t>{});

    auto Y = torch::empty_like(tY, tY.options().dtype(out_dtype));

    // For now, only support half precision input FP64 output 
    launch_diag_unscale2<at::Half,double,double,true>(tY, Y, theta, sizes, sizes_prod, r_inv);
    //launch_diag_unscale2<double,double,double,true>(tY, Y, theta, sizes, sizes_prod, r_inv);

    return Y;
}

torch::Tensor diag_unscale_forward(
    torch::Tensor tY,                 // \tilde{Y} (contiguous recommended)
    std::vector<torch::Tensor> r_inv, // length N, each 1D on device
    double theta,                     // scalar used to scale output tensor
    bool use_fp64_accum,              // Accumulate in FP64?
    c10::ScalarType out_dtype         // kFloat or kDouble, etc.
){
    const int N = tY.dim();
    TORCH_CHECK((int)r_inv.size() == N);

    // Build sizes and strides in "elements"
    std::vector<int64_t> sizes(N), strides_elts(N);
    for (int d = 0; d < N; ++d) {
        sizes[d] = tY.size(d);
        strides_elts[d] = tY.stride(d);
        TORCH_CHECK(r_inv[d].device().is_cuda());
        TORCH_CHECK(r_inv[d].numel() == sizes[d]);
    }

    // Output
    auto Y = torch::empty_like(tY, tY.options().dtype(out_dtype));

    // Dispatch combos (InT = at::Half/float/double)
    if (tY.scalar_type() == at::kHalf && out_dtype == at::kFloat) {
        if (use_fp64_accum) launch_diag_unscale<at::Half,float,double,true>(tY, Y, theta,sizes, strides_elts, r_inv);
        else                launch_diag_unscale<at::Half,float,float ,false>(tY, Y, theta, sizes, strides_elts, r_inv);
    } else if (tY.scalar_type() == at::kHalf && out_dtype == at::kHalf) {
        // Not recommended, but supported
        if (use_fp64_accum) launch_diag_unscale<at::Half,at::Half,double,true>(tY, Y, theta, sizes, strides_elts, r_inv);
        else                launch_diag_unscale<at::Half,at::Half,float ,false>(tY, Y, theta, sizes, strides_elts, r_inv);
    } else if (tY.scalar_type() == at::kFloat && out_dtype == at::kFloat) {
        if (use_fp64_accum) launch_diag_unscale<float,float,double,true>(tY, Y, theta, sizes, strides_elts, r_inv);
        else                launch_diag_unscale<float,float,float ,false>(tY, Y, theta, sizes, strides_elts, r_inv);
    } else if (tY.scalar_type() == at::kFloat && out_dtype == at::kDouble) {
        launch_diag_unscale<float,double,double,true>(tY, Y, theta, sizes, strides_elts, r_inv);
    } 
    else if (tY.scalar_type() == at::kHalf && out_dtype == at::kDouble) {
        launch_diag_unscale<at::Half,double,double,true>(tY, Y, theta, sizes, strides_elts, r_inv);
    } 
    else if (tY.scalar_type() == at::kDouble && out_dtype == at::kDouble) {
        launch_diag_unscale<double,double,double,true>(tY, Y, theta, sizes, strides_elts, r_inv);
    }
    else {
        TORCH_CHECK(false, "Unsupported dtype combination.");
    }
    return Y;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  m.def("diag_unscale_forward", &diag_unscale_forward, "Materialize Y = tY * Π r_n^{-1}");
  m.def("diag_unscale_forward2", &diag_unscale_forward2, "Materialize Y = tY * Π r_n^{-1}");
}

