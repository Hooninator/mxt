#include "hip/hip_runtime.h"
#include "mxt.cuh"

using namespace mxt;

int main(int argc, char ** argv)
{
    static constexpr uint32_t order = 3;
    using SparseTensor_t = SparseTensor<order, double, uint64_t>;

    SparseTensor_t::Index modes{6, 3, 3};

    SparseTensor_t X = io::read_tensor_frostt<SparseTensor_t>("$SCRATCH/tensors/matmul_6-3-3.tns", modes);

    SparseTensor_t::Index tucker_ranks{2, 2, 2};
    auto tucker_X = mixed_sparse_hooi<SparseTensor_t, __half, float>(X, tucker_ranks, "randn", 100);

    return 0;
}
