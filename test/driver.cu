#include "hip/hip_runtime.h"

#include "mxt.cuh"

#include <map>
#include <string>

using namespace mxt;

template <typename InputModes, typename TuckerRanks, typename HighU, typename LowU, typename LraU, typename Idx>
struct Config
{
    using InputModes_t = InputModes;
    using TuckerRanks_t = TuckerRanks;
    using HighU_t = HighU;
    using LowU_t = LowU;
    using LraU_t = LraU;
    using Idx_t = Idx;

    static constexpr uint32_t Order = InputModes_t::dims.size();
};


template <typename Conf>
void run_tensor(std::string& path)
{
    using SparseTensor_t = SparseTensor<typename Conf::HighU_t, typename Conf::Idx_t, Conf::Order, typename Conf::InputModes_t>;
    SparseTensor_t X = io::read_tensor_frostt<SparseTensor_t>(path.c_str());
    auto tucker_X = mixed_sparse_hooi<SparseTensor_t, typename Conf::TuckerRanks_t, typename Conf::LowU_t, typename Conf::LraU_t>(X, "randn", 100);
}


using NipsTns = Config<Shape<2482, 2862, 14036, 17>, 
                        Shape<10, 10, 10, 10>,
                        double, __half, float,
                        uint64_t>;


int main(int argc, char ** argv)
{
    if (argc < 3)
    {
        std::cerr<<"Usage: ./driver <tensor_name> <path_to_tensor>"<<std::endl;
        std::abort();
    }

    std::string tensor = std::string(argv[1]);
    std::string path = std::string(argv[2]);

    if (tensor.compare("nips")==0)
    {
        run_tensor<NipsTns>(path);
    }
    else
    {
        std::cerr<<"Invalid tensor: "<<tensor<<std::endl;
        std::abort();
    }
}
