#include "hip/hip_runtime.h"

#include "mxt.cuh"

#include <map>
#include <string>

using namespace mxt;

template <typename InputModes, typename TuckerRanks, typename HighU, typename LowU, typename LraU, typename Idx>
struct Config
{
    using InputModes_t = InputModes;
    using TuckerRanks_t = TuckerRanks;
    using HighU_t = HighU;
    using LowU_t = LowU;
    using LraU_t = LraU;
    using Idx_t = Idx;

    static constexpr uint32_t Order = InputModes_t::dims.size();
};


template <typename Conf>
void run_tensor(std::string& path)
{
    using SparseTensor_t = SparseTensor<typename Conf::HighU_t, typename Conf::Idx_t, Conf::Order, typename Conf::InputModes_t>;

    utils::print_separator("Beginning IO");
    SparseTensor_t X = io::read_tensor_frostt<SparseTensor_t>(path.c_str());
    utils::print_separator("Done IO");

    utils::print_separator("Beginning Tucker");
    auto tucker_X = mixed_sparse_hooi<SparseTensor_t, typename Conf::TuckerRanks_t, typename Conf::LowU_t, typename Conf::LraU_t>(X, "randn", 3);
    utils::print_separator("Done Tucker");
}


using NipsTns = Config<Shape<2482, 2862, 14036, 17>, 
                        Shape<10, 10, 10, 10>,
                        double, __half, float,
                        uint64_t>;

using Randn5Tns = Config<Shape<10, 20, 10, 5, 10>, 
                        Shape<5, 3, 3, 2, 5>,
                        double, __half, float,
                        uint64_t>;

using Randn4Tns = Config<Shape<10, 20, 20, 10>, 
                        Shape<5, 3, 3, 2>,
                        double, __half, float,
                        uint64_t>;

using Randn3Tns = Config<Shape<10, 20, 10>, 
                        Shape<5, 3, 3>,
                        double, __half, float,
                        uint64_t>;


int main(int argc, char ** argv)
{
    if (argc < 3)
    {
        std::cerr<<"Usage: ./driver <tensor_name> <path_to_tensor>"<<std::endl;
        std::abort();
    }

    std::string tensor = std::string(argv[1]);
    std::string path = std::string(argv[2]);

    if (tensor.compare("nips")==0)
    {
        run_tensor<NipsTns>(path);
    }
    else if (tensor.compare("randn3")==0)
    {
        run_tensor<Randn3Tns>(path);
    }
    else if (tensor.compare("randn4")==0)
    {
        run_tensor<Randn4Tns>(path);
    }
    else if (tensor.compare("randn5")==0)
    {
        run_tensor<Randn5Tns>(path);
    }
    else
    {
        std::cerr<<"Invalid tensor: "<<tensor<<std::endl;
        std::abort();
    }
}
