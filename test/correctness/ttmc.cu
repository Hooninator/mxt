#include "mxt.cuh"
#include "TtmcConfig.cuh"

#include <map>
#include <string>
#include <sstream>
#include <thrust/sort.h>
#include <thrust/device_ptr.h>


using namespace mxt;
static const char * base = "../test/correctness/ttmc_golden/";


template <typename Conf>
void run_correctness(std::string& path, std::string& tensorname)
{
    using DenseTensor_t = DenseTensor<typename Conf::HighU_t, typename Conf::MatrixCols_t>;
    using MatrixCollection_t = MatrixCollection<typename Conf::LowU_t, typename Conf::MatrixRows_t, typename Conf::MatrixCols_t>;
    using OutputDenseTensor_t = DenseTensor<typename DenseTensor_t::ValueType_t, typename Conf::MatrixRows_t>;

    utils::print_separator("Beginning IO");
    DenseTensor_t X(path.c_str()); 
    utils::print_separator("Done IO");

    std::string golden_dir = std::string(base);
    golden_dir.append(tensorname);
    golden_dir.append("/");

    MatrixCollection_t matrices(golden_dir.c_str());

    utils::print_separator("Beginning TTMc");
    OutputDenseTensor_t Y = ttmc_mixed<DenseTensor_t, MatrixCollection_t, OutputDenseTensor_t>(X, matrices);
    utils::print_separator("Done TTMc");

    /* Correctness check */
    std::string correct_output = golden_dir + "output.dns";
    OutputDenseTensor_t Y_correct(correct_output.c_str());
    bool correct = Y == Y_correct;

    if (correct)
    {
        std::cout<<GREEN<<"Correctness passed!"<<RESET<<std::endl;
    }
    else
    {
        std::cout<<RED<<"Correctness failed"<<RESET<<std::endl;
    }
}



using SmallDense = TtmcConfig<Shape<3, 3, 3>, 
                     Shape<2,2,2>, 
                     double, double, HIPBLAS_COMPUTE_64F, GEN_RANDN>;


using IndianPines = TtmcConfig<Shape<145, 145, 200>, 
                        Shape<20, 20, 20>,
                        double, double, HIPBLAS_COMPUTE_64F, GEN_RANDN>;


int main(int argc, char ** argv)
{
    if (argc < 2)
    {
        std::cerr<<"Usage: ./correctness <tensor_name>"<<std::endl;
        std::abort();
    }

    std::string tensor = std::string(argv[1]);
    std::stringstream ss;
    ss<<"../tensors/"<<tensor<<".dns";
    std::string path = ss.str();

    mxt_init();

    if (tensor.compare("small_dense")==0)
    {
        run_correctness<SmallDense>(path, tensor);
    }
    else if (tensor.compare("indian_pines")==0)
    {
        run_correctness<IndianPines>(path, tensor);
    }
    else
    {
        std::cerr<<"Invalid tensor: "<<tensor<<std::endl;
        std::abort();
    }

    mxt_finalize();

    return 0;
}
