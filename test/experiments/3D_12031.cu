#include "hip/hip_runtime.h"
#include "run.cuh"
using Conf0 = Config<Shape<100,80,60>,
                                                      Shape<10,8,6>,
                                                      double, double, double, double,
                                                      uint64_t>;

                using Conf1 = Config<Shape<100,80,60>,
                                                      Shape<10,8,6>,
                                                      double, float, float, float,
                                                      uint64_t>;

                using Conf2 = Config<Shape<100,80,60>,
                                                      Shape<10,8,6>,
                                                      double, __half, float, float,
                                                      uint64_t>;

                
            int main(int argc, char ** argv)
            {
                std::string path("../tensors/3D_12031.tns");
                mxt_init();
                run<Conf0,Conf1,Conf2>(path);
                mxt_finalize();
                return 0;
            }

            