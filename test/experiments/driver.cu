#include "hip/hip_runtime.h"

#include "mxt.cuh"
#include "Config.hpp"

#include <map>
#include <string>

using namespace mxt;


template <typename Conf>
void run_trial(std::string& path)
{
    using SparseTensor_t = SparseTensor<typename Conf::HighU_t, typename Conf::LowU_t, typename Conf::Idx_t, Conf::Order, typename Conf::InputModes_t>;

    utils::print_separator("Beginning IO");
    SparseTensor_t X = io::read_tensor_frostt<SparseTensor_t>(path.c_str());
    utils::print_separator("Done IO");



    utils::print_separator("Beginning Tucker");
    globals::profiler->start_timer("hooi");
    auto tucker_X = mixed_sparse_hooi<SparseTensor_t, typename Conf::CoreTensorU_t, typename Conf::LraU_t, typename Conf::TuckerRanks_t>(X, "randn", 5);
    globals::profiler->stop_timer("hooi");
    globals::profiler->print_timer("hooi");
    utils::print_separator("Done Tucker");


    auto err = tucker_X.reconstruction_error(X);
    std::cout<<"||X - X_tucker||_F / ||X||_F : "<<err<<std::endl;

    std::ofstream core_file;
    core_file.open("core.out");
    tucker_X.dump_core(core_file);
    core_file.close();
}


template <typename Conf>
void run(std::string& path)
{
    for (uint32_t t = 0; t < 1; t++)
    {
        run_trial<Conf>(path);
    }
}


using NipsTns = Config<Shape<2482, 2862, 14036, 17>, 
                        Shape<10, 10, 10, 10>,
                        double, double, double, double,
                        uint64_t>;

using ChicagoCrime = Config<Shape<6186, 24, 77, 32>,
                            Shape<20, 20, 20, 20>,
                            double, double, double, double,
                            uint64_t>;

using Randn5Tns = Config<Shape<10, 20, 10, 5, 10>, 
                        Shape<5, 3, 3, 2, 5>,
                        double, double, double,float,
                        uint64_t>;

using Randn4Tns = Config<Shape<10, 20, 20, 10>, 
                        Shape<5, 3, 3, 2>,
                        double, __half, float, float,
                        uint64_t>;

using Randn3Tns = Config<Shape<10, 20, 10>, 
                        Shape<5, 3, 3>,
                        double, __half, float, float,
                        uint64_t>;


int main(int argc, char ** argv)
{
    if (argc < 3)
    {
        std::cerr<<"Usage: ./driver <tensor_name> <path_to_tensor>"<<std::endl;
        std::abort();
    }

    std::string tensor = std::string(argv[1]);
    std::string path = std::string(argv[2]);

    mxt_init();

    if (tensor.compare("nips")==0)
    {
        run<NipsTns>(path);
    }
    else if (tensor.compare("crime")==0)
    {
        run<ChicagoCrime>(path);
    }
    else if (tensor.compare("randn3")==0)
    {
        run<Randn3Tns>(path);
    }
    else if (tensor.compare("randn4")==0)
    {
        run<Randn4Tns>(path);
    }
    else if (tensor.compare("randn5")==0)
    {
        run<Randn5Tns>(path);
    }
    else
    {
        std::cerr<<"Invalid tensor: "<<tensor<<std::endl;
        std::abort();
    }

    mxt_finalize();

    return 0;
}
