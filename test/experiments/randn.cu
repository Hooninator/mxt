#include "hip/hip_runtime.h"
#include "run_ttmc.cuh"
using Conf0 = TtmcConfig<Shape<100,100,100,100>,
                                                      Shape<50,50,50,50>,
                                                      double, double, HIPBLAS_COMPUTE_64F>;

                using Conf1 = TtmcConfig<Shape<100,100,100,100>,
                                                      Shape<50,50,50,50>,
                                                      double, float, HIPBLAS_COMPUTE_32F>;

                using Conf2 = TtmcConfig<Shape<100,100,100,100>,
                                                      Shape<50,50,50,50>,
                                                      double, float, HIPBLAS_COMPUTE_32F_FAST_16F>;

                using Conf3 = TtmcConfig<Shape<100,100,100,100>,
                                                      Shape<50,50,50,50>,
                                                      double, __half, HIPBLAS_COMPUTE_16F>;

                
            int main(int argc, char ** argv)
            {
                std::string path("../tensors/randn.dns");
                mxt_init();
                run<Conf0,Conf1,Conf2,Conf3>(path);
                mxt_finalize();
                return 0;
            }

            